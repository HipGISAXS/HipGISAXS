#include "hip/hip_runtime.h"
/***
  *  Project: HipGISAXS (High-Performance GISAXS)
  *
  *  File: ff_ana_cylinder_gpu.cu
  *  Created: Oct 16, 2012
  *  Modified: Sat 02 Mar 2013 11:40:02 AM PST
  *
  *  Author: Abhinav Sarje <asarje@lbl.gov>
  */

#include <iostream>
#include <complex>
#include <hip/hip_complex.h>

#include "ff_ana_gpu.cuh"
#include "enums.hpp"
#include "cu_complex_numeric.cuh"
#include "cu_utilities.cuh"

namespace hig {

	/**
	 * horizontal cylinder on gpu
	 */

	__global__ void form_factor_hcylinder_kernel(unsigned int, unsigned int, unsigned int,
									float_t*, float_t*, cucomplex_t*, float_t, float_t, float_t*,
									unsigned int, float_t*, unsigned int, float_t*,
									unsigned int, float_t*, unsigned int, float_t*,
									float_t*, cucomplex_t*);


	bool AnalyticFormFactorG::compute_horizontal_cylinder(const float_t tau, const float_t eta,
									const std::vector<float_t>& h,
									const std::vector<float_t>& distr_h,
									const std::vector<float_t>& r,
									const std::vector<float_t>& distr_r,
									const float_t* rot, const std::vector<float_t>& transvec,
									std::vector<complex_t>& ff) {
		unsigned int n_h = h.size(), n_distr_h = distr_h.size();
		unsigned int n_r = r.size(), n_distr_r = distr_r.size();
		const float_t *h_h = h.empty() ? NULL : &*h.begin();
		const float_t *distr_h_h = distr_h.empty() ? NULL : &*distr_h.begin();
		const float_t *r_h = r.empty() ? NULL : &*r.begin();
		const float_t *distr_r_h = distr_r.empty() ? NULL : &*distr_r.begin();

		run_init(rot, transvec);

		// construct device buffers
		float_t *h_d, *distr_h_d;
		float_t *r_d, *distr_r_d;

		hipMalloc((void**) &h_d, n_h * sizeof(float_t));
		hipMalloc((void**) &distr_h_d, n_distr_h * sizeof(float_t));
		hipMalloc((void**) &r_d, n_r * sizeof(float_t));
		hipMalloc((void**) &distr_r_d, n_distr_r * sizeof(float_t));

		// copy data to device buffers
		hipMemcpy(h_d, h_h, n_h * sizeof(float_t), hipMemcpyHostToDevice);
		hipMemcpy(r_d, r_h, n_r * sizeof(float_t), hipMemcpyHostToDevice);
		hipMemcpy(distr_h_d, distr_h_h, n_distr_h * sizeof(float_t), hipMemcpyHostToDevice);
		hipMemcpy(distr_r_d, distr_r_h, n_distr_r * sizeof(float_t), hipMemcpyHostToDevice);

		unsigned int cuda_block_y = 16, cuda_block_z = 8;
		unsigned int cuda_num_blocks_y = (unsigned int) ceil((float_t) nqy_ / cuda_block_y);
		unsigned int cuda_num_blocks_z = (unsigned int) ceil((float_t) nqz_ / cuda_block_z);
		dim3 ff_grid_size(cuda_num_blocks_y, cuda_num_blocks_z, 1);
		dim3 ff_block_size(cuda_block_y, cuda_block_z, 1);

		// the kernel
		form_factor_hcylinder_kernel <<< ff_grid_size, ff_block_size >>> (
					nqx_, nqy_, nqz_, qx_, qy_, qz_, tau, eta, rot_,
					n_h, h_d, n_distr_h, distr_h_d,
					n_r, r_d, n_distr_r, distr_r_d,
					transvec_,
					ff_);

		hipDeviceSynchronize();
		hipError_t err = hipGetLastError();
		if(err != hipSuccess) {
			std::cerr << "error: box form factor kernel failed [" << __FILE__ << ":" << __LINE__ << "]: "
						<< hipGetErrorString(err) << std::endl;
			return false;
		} else {
			//std::cout << "block size: " << cby << " x " << cbz << ". ";
			construct_output_ff(ff);
		} // if-else

		hipFree(distr_r_d);
		hipFree(r_d);
		hipFree(distr_h_d);
		hipFree(h_d);

		return true;
	} // AnalyticFormFactorG::compute_horizontal_cylinder()


	__global__ void form_factor_hcylinder_kernel(unsigned int nqx, unsigned int nqy, unsigned int nqz,
									float_t *qx, float_t *qy, cucomplex_t *qz,
									float_t tau, float_t eta, float_t *rot,
									unsigned int n_h, float_t *h, unsigned int n_distr_h, float_t *distr_h,
									unsigned int n_r, float_t *r, unsigned int n_distr_r, float_t *distr_r,
									float_t *transvec, cucomplex_t *ff) {
		unsigned int i_y = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int i_z = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int base_index = nqx * nqy * i_z + nqx * i_y;
		if(i_y < nqy && i_z < nqz) {
			for(unsigned int i_x = 0; i_x < nqx; ++ i_x) {
				cucomplex_t mqx, mqy, mqz;
				compute_meshpoints(qx[i_x], qy[i_y], qz[i_z], rot, mqx, mqy, mqz);
				cucomplex_t qpar = cuCsqrt(mqz * mqz + mqy * mqy);
				cucomplex_t temp_ff = make_cuC((float_t) 0.0, (float_t) 0.0);
				// why does this not depend on eta? ...
				for(unsigned int p_r = 0; p_r < n_r; ++ p_r) {
					for(unsigned int p_h = 0; p_h < n_h; ++ p_h) {
						float_t temp1 = distr_r[p_r] * distr_h[p_h] * 2 * PI_ * r[p_r] * r[p_r];
						cucomplex_t temp2 = qpar * r[p_r];
						cucomplex_t temp3 = cuCcbessj(temp2, 1) / temp2;
						cucomplex_t temp4 = fq_inv(mqx, h[p_h]);
						temp_ff = temp_ff + temp1 * temp3 * temp4;
					} // for h
				} // for r
				cucomplex_t temp_e = cuCexpi(mqx * transvec[0] + mqy * transvec[1] + mqz * transvec[2]);
				unsigned int curr_index = base_index + i_x;
				ff[curr_index] = temp_ff * temp_e;
			} // for x
		} // if
	} // form_factor_hcylinder_kernel()

} // namespace hig

