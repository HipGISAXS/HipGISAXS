#include "hip/hip_runtime.h"
/***
  *  Project: HipGISAXS (High-Performance GISAXS)
  *
  *  File: ff_ana_gpu.cu
  *  Created: Oct 16, 2012
  *  Modified: Wed 20 Feb 2013 01:06:30 PM PST
  *
  *  Author: Abhinav Sarje <asarje@lbl.gov>
  */

#include <iostream>
#include <complex>
#include <hip/hip_complex.h>

#include "ff_ana_gpu.cuh"
#include "enums.hpp"
#include "cu_complex_numeric.cuh"
#include "cu_utilities.cuh"


namespace hig {

	/**
	 * box on gpu
	 */

	__global__ void form_factor_box_kernel(unsigned int, unsigned int, unsigned int,
									float_t*, float_t*, cucomplex_t*, float_t, float_t, float_t*,
									unsigned int, float_t*, unsigned int, float_t*,
									unsigned int, float_t*, unsigned int, float_t*,
									unsigned int, float_t*, unsigned int, float_t*,
									/*unsigned int,*/ float_t*, cucomplex_t*);


	bool AnalyticFormFactorG::compute_box(const float_t tau, const float_t eta,
										const std::vector<float_t>& x,
										const std::vector<float_t>& distr_x,
										const std::vector<float_t>& y,
										const std::vector<float_t>& distr_y,
										const std::vector<float_t>& z,
										const std::vector<float_t>& distr_z,
										/*const float_t* qx_h, const float_t* qy_h,
										const cucomplex_t* qz_h,*/
										const float_t* rot_h, const std::vector<float_t>& transvec,
										std::vector<complex_t>& ff) {
		unsigned int n_x = x.size(), n_distr_x = distr_x.size();
		unsigned int n_y = y.size(), n_distr_y = distr_y.size();
		unsigned int n_z = z.size(), n_distr_z = distr_z.size();
		//unsigned int n_transvec = transvec.size();
		const float_t *x_h = x.empty() ? NULL : &*x.begin();
		const float_t *distr_x_h = distr_x.empty() ? NULL : &*distr_x.begin();
		const float_t *y_h = y.empty() ? NULL : &*y.begin();
		const float_t *distr_y_h = distr_y.empty() ? NULL : &*distr_y.begin();
		const float_t *z_h = z.empty() ? NULL : &*z.begin();
		const float_t *distr_z_h = distr_z.empty() ? NULL : &*distr_z.begin();
		//const float_t *transvec_h = transvec.empty() ? NULL : &*transvec.begin();

		unsigned int grid_size = nqx_ * nqy_ * nqz_;

		// construct device buffers
		//float_t *qx_d, *qy_d;
		//cucomplex_t *qz_d, *ff_d;
		float_t *x_d, *distr_x_d;
		float_t *y_d, *distr_y_d;
		float_t *z_d, *distr_z_d;
		//float_t *transvec_d, *rot_d;

		//hipMalloc((void**) &qx_d, nqx_ * sizeof(float_t));
		//hipMalloc((void**) &qy_d, nqy_ * sizeof(float_t));
		//hipMalloc((void**) &qz_d, nqz_ * sizeof(cucomplex_t));
		//hipMalloc((void**) &ff_d, grid_size * sizeof(cucomplex_t));
		hipMalloc((void**) &x_d, n_x * sizeof(float_t));
		hipMalloc((void**) &distr_x_d, n_distr_x * sizeof(float_t));
		hipMalloc((void**) &y_d, n_y * sizeof(float_t));
		hipMalloc((void**) &distr_y_d, n_distr_y * sizeof(float_t));
		hipMalloc((void**) &z_d, n_z * sizeof(float_t));
		hipMalloc((void**) &distr_z_d, n_distr_z * sizeof(float_t));
		//hipMalloc((void **) &transvec_d, n_transvec * sizeof(float_t));
		//hipMalloc((void **) &rot_d, 9 * sizeof(float_t));

		// copy data to device buffers
		//hipMemcpy(qx_d, qx_h, nqx_ * sizeof(float_t), hipMemcpyHostToDevice);
		//hipMemcpy(qy_d, qy_h, nqy_ * sizeof(float_t), hipMemcpyHostToDevice);
		//hipMemcpy(qz_d, qz_h, nqz_ * sizeof(cucomplex_t), hipMemcpyHostToDevice);
		hipMemcpy(x_d, x_h, n_x * sizeof(float_t), hipMemcpyHostToDevice);
		hipMemcpy(y_d, y_h, n_y * sizeof(float_t), hipMemcpyHostToDevice);
		hipMemcpy(z_d, z_h, n_z * sizeof(float_t), hipMemcpyHostToDevice);
		hipMemcpy(distr_x_d, distr_x_h, n_distr_x * sizeof(float_t), hipMemcpyHostToDevice);
		hipMemcpy(distr_y_d, distr_y_h, n_distr_y * sizeof(float_t), hipMemcpyHostToDevice);
		hipMemcpy(distr_z_d, distr_z_h, n_distr_z * sizeof(float_t), hipMemcpyHostToDevice);
		//hipMemcpy(transvec_d, transvec_h, n_transvec * sizeof(float_t), hipMemcpyHostToDevice);
		//hipMemcpy(rot_d, rot_h, 9 * sizeof(float_t), hipMemcpyHostToDevice);

		run_init(rot_h, transvec);

		unsigned int cuda_block_y = 16, cuda_block_z = 8;
		unsigned int cuda_num_blocks_y = (unsigned int) ceil((float_t) nqy_ / cuda_block_y);
		unsigned int cuda_num_blocks_z = (unsigned int) ceil((float_t) nqz_ / cuda_block_z);
		dim3 ff_grid_size(cuda_num_blocks_y, cuda_num_blocks_z, 1);
		dim3 ff_block_size(cuda_block_y, cuda_block_z, 1);

		// the kernel
		form_factor_box_kernel <<< ff_grid_size, ff_block_size >>> (
				nqx_, nqy_, nqz_, qx_, qy_, qz_, tau, eta, rot_,
				n_x, x_d, n_distr_x, distr_x_d,
				n_y, y_d, n_distr_y, distr_y_d,
				n_z, z_d, n_distr_z, distr_z_d,
				/*n_transvec,*/ transvec_,
				ff_);

		hipDeviceSynchronize();
		hipError_t err = hipGetLastError();
		if(err != hipSuccess) {
			std::cerr << "error: box form factor kernel failed [" << __FILE__ << ":" << __LINE__ << "]: "
						<< hipGetErrorString(err) << std::endl;
			return false;
		} else {
			//std::cout << "block size: " << cby << " x " << cbz << ". ";
			/*cucomplex_t* ff_h = new (std::nothrow) cucomplex_t[grid_size];
			// copy result to host
			hipMemcpy(ff_h, ff_, grid_size * sizeof(cucomplex_t), hipMemcpyDeviceToHost);
			ff.clear();
			ff.reserve(grid_size);
			for(unsigned int i = 0; i < grid_size; ++ i) {
				ff.push_back(complex_t(ff_h[i].x, ff_h[i].y));
			} // for
			delete[] ff_h;*/
			construct_output_ff(ff);
		} // if-else

		//hipFree(rot_d);
		//hipFree(transvec_d);
		hipFree(distr_z_d);
		hipFree(z_d);
		hipFree(distr_y_d);
		hipFree(y_d);
		hipFree(distr_x_d);
		hipFree(x_d);
		//hipFree(ff_d);
		//hipFree(qz_d);
		//hipFree(qy_d);
		//hipFree(qx_d);

		return true;
	} // AnalyticFormFactorG::compute_box()


	__global__ void form_factor_box_kernel(unsigned int nqx, unsigned int nqy, unsigned int nqz,
									float_t *qx, float_t *qy, cucomplex_t *qz,
									float_t tau, float_t eta, float_t *rot,
									unsigned int n_x, float_t *x, unsigned int n_distr_x, float_t *distr_x,
									unsigned int n_y, float_t *y, unsigned int n_distr_y, float_t *distr_y,
									unsigned int n_z, float_t *z, unsigned int n_distr_z, float_t *distr_z,
									/*unsigned int n_transvec,*/ float_t *transvec, cucomplex_t *ff) {
		unsigned int i_y = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int i_z = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int base_index = nqx * nqy * i_z + nqx * i_y;
		if(i_y < nqy && i_z < nqz) {
			for(unsigned int i_x = 0; i_x < nqx; ++ i_x) {
				cucomplex_t mqx = make_cuC(qy[i_y] * rot[0] + qx[i_x] * rot[1] + qz[i_z].x * rot[2],
											qz[i_z].y * rot[2]);
				cucomplex_t mqy = make_cuC(qy[i_y] * rot[3] + qx[i_x] * rot[4] + qz[i_z].x * rot[5],
											qz[i_z].y * rot[5]);
				cucomplex_t mqz = make_cuC(qy[i_y] * rot[6] + qx[i_x] * rot[7] + qz[i_z].x * rot[8],
											qz[i_z].y * rot[8]);
				cucomplex_t tempa = sin(eta) * mqx;
				cucomplex_t tempb = cos(eta) * mqy;
				cucomplex_t temp_qm = tan(tau) * (tempa + tempb);
				cucomplex_t temp_ff = make_cuC((float_t) 0.0, (float_t) 0.0);
				for(unsigned int p_z = 0; p_z < n_z; ++ p_z) {
					for(unsigned int p_y = 0; p_y < n_y; ++ p_y) {
						for(unsigned int p_x = 0; p_x < n_x; ++ p_x) {
							cucomplex_t temp4 = fq_inv(mqz + temp_qm, y[p_y]);
							cucomplex_t temp8 = temp4 * sinc(mqy * z[p_z]) * sinc(mqx * x[p_x]);
							float_t temp9 = 4.0 * distr_x[p_x] * distr_y[p_y] * distr_z[p_z] *
											z[p_z] * x[p_x];
							temp_ff = temp_ff + temp9 * temp8;
						} // for x
					} // for y
				} // for z
				cucomplex_t temp_e = cuCexp(mqx * transvec[0] + mqy * transvec[1] + mqz * transvec[2]);
				unsigned int curr_index = base_index + i_x;
				ff[curr_index] = temp_ff * temp_e;
			} // for x
		} // if
	} // form_factor_box_kernel()


} // namespace hig

