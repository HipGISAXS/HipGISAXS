#include "hip/hip_runtime.h"
/***
  *  Project: HipGISAXS (High-Performance GISAXS)
  *
  *  File: ff_ana_gpu.cu
  *  Created: Oct 16, 2012
  *  Modified: Fri 22 Feb 2013 09:45:21 AM PST
  *
  *  Author: Abhinav Sarje <asarje@lbl.gov>
  */

#include <iostream>
#include <complex>
#include <hip/hip_complex.h>

#include "ff_ana_gpu.cuh"
#include "enums.hpp"
#include "cu_complex_numeric.cuh"
#include "cu_utilities.cuh"


namespace hig {

	/**
	 * box on gpu
	 */

	__global__ void form_factor_box_kernel(unsigned int, unsigned int, unsigned int,
									float_t*, float_t*, cucomplex_t*, float_t, float_t, float_t*,
									unsigned int, float_t*, unsigned int, float_t*,
									unsigned int, float_t*, unsigned int, float_t*,
									unsigned int, float_t*, unsigned int, float_t*,
									float_t*, cucomplex_t*);


	bool AnalyticFormFactorG::compute_box(const float_t tau, const float_t eta,
									const std::vector<float_t>& x,
									const std::vector<float_t>& distr_x,
									const std::vector<float_t>& y,
									const std::vector<float_t>& distr_y,
									const std::vector<float_t>& z,
									const std::vector<float_t>& distr_z,
									const float_t* rot_h, const std::vector<float_t>& transvec,
									std::vector<complex_t>& ff) {
		unsigned int n_x = x.size(), n_distr_x = distr_x.size();
		unsigned int n_y = y.size(), n_distr_y = distr_y.size();
		unsigned int n_z = z.size(), n_distr_z = distr_z.size();
		const float_t *x_h = x.empty() ? NULL : &*x.begin();
		const float_t *distr_x_h = distr_x.empty() ? NULL : &*distr_x.begin();
		const float_t *y_h = y.empty() ? NULL : &*y.begin();
		const float_t *distr_y_h = distr_y.empty() ? NULL : &*distr_y.begin();
		const float_t *z_h = z.empty() ? NULL : &*z.begin();
		const float_t *distr_z_h = distr_z.empty() ? NULL : &*distr_z.begin();

		unsigned int grid_size = nqx_ * nqy_ * nqz_;

		// construct device buffers
		float_t *x_d, *distr_x_d;
		float_t *y_d, *distr_y_d;
		float_t *z_d, *distr_z_d;

		hipMalloc((void**) &x_d, n_x * sizeof(float_t));
		hipMalloc((void**) &distr_x_d, n_distr_x * sizeof(float_t));
		hipMalloc((void**) &y_d, n_y * sizeof(float_t));
		hipMalloc((void**) &distr_y_d, n_distr_y * sizeof(float_t));
		hipMalloc((void**) &z_d, n_z * sizeof(float_t));
		hipMalloc((void**) &distr_z_d, n_distr_z * sizeof(float_t));

		// copy data to device buffers
		hipMemcpy(x_d, x_h, n_x * sizeof(float_t), hipMemcpyHostToDevice);
		hipMemcpy(y_d, y_h, n_y * sizeof(float_t), hipMemcpyHostToDevice);
		hipMemcpy(z_d, z_h, n_z * sizeof(float_t), hipMemcpyHostToDevice);
		hipMemcpy(distr_x_d, distr_x_h, n_distr_x * sizeof(float_t), hipMemcpyHostToDevice);
		hipMemcpy(distr_y_d, distr_y_h, n_distr_y * sizeof(float_t), hipMemcpyHostToDevice);
		hipMemcpy(distr_z_d, distr_z_h, n_distr_z * sizeof(float_t), hipMemcpyHostToDevice);

		run_init(rot_h, transvec);

		unsigned int cuda_block_y = 16, cuda_block_z = 8;
		unsigned int cuda_num_blocks_y = (unsigned int) ceil((float_t) nqy_ / cuda_block_y);
		unsigned int cuda_num_blocks_z = (unsigned int) ceil((float_t) nqz_ / cuda_block_z);
		dim3 ff_grid_size(cuda_num_blocks_y, cuda_num_blocks_z, 1);
		dim3 ff_block_size(cuda_block_y, cuda_block_z, 1);

		size_t shared_mem_size = (nqx_ + cuda_block_y) * sizeof(float_t) +
									cuda_block_z * sizeof(cucomplex_t);
		if(shared_mem_size > 49152) {
			std::cerr << "Too much shared memory requested!" << std::endl;
			return false;
		} // if

		// the kernel
		//form_factor_box_kernel <<< ff_grid_size, ff_block_size >>> (
		form_factor_box_kernel <<< ff_grid_size, ff_block_size, shared_mem_size >>> (
				nqx_, nqy_, nqz_, qx_, qy_, qz_, tau, eta, rot_,
				n_x, x_d, n_distr_x, distr_x_d,
				n_y, y_d, n_distr_y, distr_y_d,
				n_z, z_d, n_distr_z, distr_z_d,
				transvec_,
				ff_);

		hipDeviceSynchronize();
		hipError_t err = hipGetLastError();
		if(err != hipSuccess) {
			std::cerr << "error: box form factor kernel failed [" << __FILE__ << ":" << __LINE__ << "]: "
						<< hipGetErrorString(err) << std::endl;
			return false;
		} else {
			//std::cout << "block size: " << cby << " x " << cbz << ". ";
			construct_output_ff(ff);
		} // if-else

		hipFree(distr_z_d);
		hipFree(z_d);
		hipFree(distr_y_d);
		hipFree(y_d);
		hipFree(distr_x_d);
		hipFree(x_d);

		return true;
	} // AnalyticFormFactorG::compute_box()


/*	__global__ void form_factor_box_kernel(unsigned int nqx, unsigned int nqy, unsigned int nqz,
									float_t *qx, float_t *qy, cucomplex_t *qz,
									float_t tau, float_t eta, float_t *rot,
									unsigned int n_x, float_t *x, unsigned int n_distr_x, float_t *distr_x,
									unsigned int n_y, float_t *y, unsigned int n_distr_y, float_t *distr_y,
									unsigned int n_z, float_t *z, unsigned int n_distr_z, float_t *distr_z,
									float_t *transvec, cucomplex_t *ff) {
		unsigned int i_y = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int i_z = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int base_index = nqx * nqy * i_z + nqx * i_y;
		if(i_y < nqy && i_z < nqz) {
			for(unsigned int i_x = 0; i_x < nqx; ++ i_x) {
				cucomplex_t mqx, mqy, mqz;
				compute_meshpoints(qx[i_x], qy[i_y], qz[i_z], rot, mqx, mqy, mqz);
				cucomplex_t tempa = sin(eta) * mqx;
				cucomplex_t tempb = cos(eta) * mqy;
				cucomplex_t temp_qm = tan(tau) * (tempa + tempb);
				cucomplex_t temp_ff = make_cuC((float_t) 0.0, (float_t) 0.0);
				for(unsigned int p_z = 0; p_z < n_z; ++ p_z) {
					for(unsigned int p_y = 0; p_y < n_y; ++ p_y) {
						for(unsigned int p_x = 0; p_x < n_x; ++ p_x) {
							cucomplex_t temp4 = fq_inv(mqz + temp_qm, y[p_y]);
							cucomplex_t temp8 = temp4 * cuCsinc(mqy * z[p_z]) * cuCsinc(mqx * x[p_x]);
							float_t temp9 = 4.0 * distr_x[p_x] * distr_y[p_y] * distr_z[p_z] *
											z[p_z] * x[p_x];
							temp_ff = temp_ff + temp9 * temp8;
						} // for x
					} // for y
				} // for z
				cucomplex_t temp_e = cuCexpi(mqx * transvec[0] + mqy * transvec[1] + mqz * transvec[2]);
				unsigned int curr_index = base_index + i_x;
				ff[curr_index] = temp_ff * temp_e;
			} // for x
		} // if
	} // form_factor_box_kernel()*/


	extern __shared__ float_t dynamic_shared[];

	__global__ void form_factor_box_kernel(unsigned int nqx, unsigned int nqy, unsigned int nqz,
									float_t *qx, float_t *qy, cucomplex_t *qz,
									float_t tau, float_t eta, float_t *rot,
									unsigned int n_x, float_t *x, unsigned int n_distr_x, float_t *distr_x,
									unsigned int n_y, float_t *y, unsigned int n_distr_y, float_t *distr_y,
									unsigned int n_z, float_t *z, unsigned int n_distr_z, float_t *distr_z,
									float_t *transvec, cucomplex_t *ff) {
		unsigned int i_y = blockDim.x * blockIdx.x + threadIdx.x;
		unsigned int i_z = blockDim.y * blockIdx.y + threadIdx.y;
		unsigned int base_index = nqx * nqy * i_z + nqx * i_y;

		// shared buffers:
		cucomplex_t* qz_s = (cucomplex_t*) dynamic_shared;
		float_t* qx_s = (float_t*) &qz_s[blockDim.y];
		float_t* qy_s = (float_t*) &qx_s[nqx];

		// load all qx
		unsigned int i_thread = blockDim.x * threadIdx.y + threadIdx.x;
		unsigned int num_threads = blockDim.x * blockDim.y;
		unsigned int num_loads = ceil((float_t) nqx / num_threads);
		for(int i = 0; i < num_loads; ++ i) {
			unsigned int index = i * num_threads + i_thread;
			if(index < nqx) qx_s[index] = qx[index];
			else ;	// nop
		} // for
		// load part of qy
		if(i_y < nqy && threadIdx.y == 0)	// first row of threads
			qy_s[threadIdx.x] = qy[i_y];
		// load part of qz
		if(i_z < nqz && threadIdx.x == 0)	// first column of threads
			qz_s[threadIdx.y] = qz[i_z];

		// TODO: also put x, y, z, distr_x, distr_y, distr_z in shared mem ...
		// TODO: also put transvec in shared mem ...

		// make sure everything is in place
		__syncthreads();

		if(i_y < nqy && i_z < nqz) {
			for(unsigned int i_x = 0; i_x < nqx; ++ i_x) {
				cucomplex_t mqx, mqy, mqz;
				compute_meshpoints(qx_s[i_x], qy_s[threadIdx.x], qz_s[threadIdx.y], rot, mqx, mqy, mqz);
				cucomplex_t tempa = sin(eta) * mqx;
				cucomplex_t tempb = cos(eta) * mqy;
				cucomplex_t temp_qm = tan(tau) * (tempa + tempb);
				cucomplex_t temp_ff = make_cuC((float_t) 0.0, (float_t) 0.0);
				for(unsigned int p_z = 0; p_z < n_z; ++ p_z) {
					for(unsigned int p_y = 0; p_y < n_y; ++ p_y) {
						for(unsigned int p_x = 0; p_x < n_x; ++ p_x) {
							cucomplex_t temp4 = fq_inv(mqz + temp_qm, y[p_y]);
							cucomplex_t temp8 = temp4 * cuCsinc(mqy * z[p_z]) * cuCsinc(mqx * x[p_x]);
							float_t temp9 = 4.0 * distr_x[p_x] * distr_y[p_y] * distr_z[p_z] *
											z[p_z] * x[p_x];
							temp_ff = temp_ff + temp9 * temp8;
						} // for x
					} // for y
				} // for z
				cucomplex_t temp_e = cuCexpi(mqx * transvec[0] + mqy * transvec[1] + mqz * transvec[2]);
				unsigned int curr_index = base_index + i_x;
				ff[curr_index] = temp_ff * temp_e;
			} // for x
		} // if
	} // form_factor_box_kernel()

} // namespace hig

