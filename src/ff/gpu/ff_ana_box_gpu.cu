#include "hip/hip_runtime.h"
/**
 *  Project: HipGISAXS (High-Performance GISAXS)
 *
 *  File: ff_ana_gpu.cu
 *  Created: Oct 16, 2012
 *  Modified: Wed 08 Oct 2014 12:17:47 PM PDT
 *
 *  Author: Abhinav Sarje <asarje@lbl.gov>
 *  Developers: Slim Chourou <stchourou@lbl.gov>
 *              Abhinav Sarje <asarje@lbl.gov>
 *              Elaine Chan <erchan@lbl.gov>
 *              Alexander Hexemer <ahexemer@lbl.gov>
 *              Xiaoye Li <xsli@lbl.gov>
 *
 *  Licensing: The HipGISAXS software is only available to be downloaded and
 *  used by employees of academic research institutions, not-for-profit
 *  research laboratories, or governmental research facilities. Please read the
 *  accompanying LICENSE file before downloading the software. By downloading
 *  the software, you are agreeing to be bound by the terms of this
 *  NON-COMMERCIAL END USER LICENSE AGREEMENT.
 */

#include <iostream>
#include <fstream>
#include <complex>
#include <hip/hip_complex.h>
#include <stdio.h>

#include <ff/gpu/ff_ana_gpu.cuh>
#include <common/enums.hpp>
#include <common/constants.hpp>
#include <numerics/gpu/cu_complex_numeric.cuh>
#include <utils/gpu/cu_utilities.cuh>


namespace hig {

  extern __constant__ real_t tau_d;
  extern __constant__ real_t eta_d;
  extern __constant__ real_t transvec_d[3];
  extern __constant__ real_t rot_d[9];

  /** Form Factor of Box:
   *  L : (real) Length of the box 
   *  W : (real) Width of the box
   *  H : (real) Height of the box
   *  q : (complex) q-vector
   *  ff = L * W * H exp(j * qz * H/2) * sinc(qx * L/2) * sinc (qy * W/2) * sinc(qz * H/2)
   */
  __device__  __inline__ cucomplex_t FormFactorBox(cucomplex_t qx, cucomplex_t qy, cucomplex_t qz, 
          real_t L, real_t W, real_t H ){
    cucomplex_t temp1 = cuCsinc (0.5 * qx * L);
    cucomplex_t temp2 = cuCsinc (0.5 * qy * W);
    cucomplex_t temp3 = cuCsinc (0.5 * qz * H);
    cucomplex_t temp4 = L * W * H * cuCexpi(0.5 * qz * H);
    return (temp1 * temp2 * temp3 * temp4);
  }
 
  __global__ void ff_box_kernel (unsigned int nqy, unsigned int nqz, 
          real_t * qx, real_t * qy, cucomplex_t * qz, cucomplex_t * ff,
          int nx, real_t * x, real_t * distr_x,
          int ny, real_t * y, real_t * distr_y,
          int nz, real_t * z, real_t * distr_z) {
    int i_z = blockDim.x * blockIdx.x + threadIdx.x;
    if (i_z < nqz){
      int i_y = i_z % nqy;
      cucomplex_t c_neg_unit = make_cuC(ZERO, NEG_ONE);
      cucomplex_t mqx, mqy, mqz;
      compute_meshpoints(qx[i_y], qy[i_y], qz[i_z], rot_d, mqx, mqy, mqz);
      cucomplex_t temp_ff = make_cuC(ZERO, ZERO);
      for (int i = 0; i < nx; i++){
        for (int j = 0; j < ny; j++){
          for (int k = 0; k < nz; k++){
            temp_ff = temp_ff + FormFactorBox(mqx, mqy, mqz, x[i], y[j], z[k]); 
          }
        }
      }
      cucomplex_t temp1 = transvec_d[0] * mqx + transvec_d[1] * mqy + transvec_d[2] * mqz;
      ff[i_z] =  temp_ff * cuCexpi(temp1);
    }
  } // ff_box_kernel()

   
  bool AnalyticFormFactorG::compute_box(const real_t tau, const real_t eta,
                  const std::vector<real_t>& x,
                  const std::vector<real_t>& distr_x,
                  const std::vector<real_t>& y,
                  const std::vector<real_t>& distr_y,
                  const std::vector<real_t>& z,
                  const std::vector<real_t>& distr_z,
                  const real_t* rot_h, const std::vector<real_t>& transvec,
                  std::vector<complex_t>& ff) {
    unsigned int n_x = x.size(), n_distr_x = distr_x.size();
    unsigned int n_y = y.size(), n_distr_y = distr_y.size();
    unsigned int n_z = z.size(), n_distr_z = distr_z.size();
    const real_t *x_h = x.empty() ? NULL : &*x.begin();
    const real_t *distr_x_h = distr_x.empty() ? NULL : &*distr_x.begin();
    const real_t *y_h = y.empty() ? NULL : &*y.begin();
    const real_t *distr_y_h = distr_y.empty() ? NULL : &*distr_y.begin();
    const real_t *z_h = z.empty() ? NULL : &*z.begin();
    const real_t *distr_z_h = distr_z.empty() ? NULL : &*distr_z.begin();
    real_t transvec_h[3] = {transvec[0], transvec[1], transvec[2]};

    // construct device buffers
    real_t *x_d, *distr_x_d;
    real_t *y_d, *distr_y_d;
    real_t *z_d, *distr_z_d;

    hipMalloc((void**) &x_d, n_x * sizeof(real_t));
    hipMalloc((void**) &distr_x_d, n_distr_x * sizeof(real_t));
    hipMalloc((void**) &y_d, n_y * sizeof(real_t));
    hipMalloc((void**) &distr_y_d, n_distr_y * sizeof(real_t));
    hipMalloc((void**) &z_d, n_z * sizeof(real_t));
    hipMalloc((void**) &distr_z_d, n_distr_z * sizeof(real_t));

    // copy data to device buffers
    hipMemcpy(x_d, x_h, n_x * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y_h, n_y * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(z_d, z_h, n_z * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(distr_x_d, distr_x_h, n_distr_x * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(distr_y_d, distr_y_h, n_distr_y * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(distr_z_d, distr_z_h, n_distr_z * sizeof(real_t), hipMemcpyHostToDevice);

    //run_init(rot_h, transvec);
    hipMemcpyToSymbol(HIP_SYMBOL(tau_d), &tau, sizeof(real_t), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(eta_d), &eta, sizeof(real_t), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(rot_d), rot_h, 9*sizeof(real_t), 0, hipMemcpyHostToDevice); 
    hipMemcpyToSymbol(HIP_SYMBOL(transvec_d), transvec_h, 3*sizeof(real_t), 0, hipMemcpyHostToDevice); 

    int num_threads = 256;
    int num_blocks =  nqz_ / num_threads + 1;
    dim3 ff_grid_size(num_blocks, 1, 1);
    dim3 ff_block_size(num_threads, 1, 1);
    std::cerr << "Q-Grid size = " << nqz_ << std::endl;

    // the kernel
    ff_box_kernel <<<num_blocks, num_threads >>> (nqy_, nqz_, 
            qx_, qy_, qz_, ff_, 
            n_x, x_d, distr_x_d, 
            n_y, y_d, distr_y_d,
            n_z, z_d, distr_z_d);
    
    construct_output_ff(ff);

    hipFree(distr_z_d);
    hipFree(z_d);
    hipFree(distr_y_d);
    hipFree(y_d);
    hipFree(distr_x_d);
    hipFree(x_d);
   
    return true;
  } // AnalyticFormFactorG::compute_box()

} // namespace hig
