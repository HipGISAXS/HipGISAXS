#include "hip/hip_runtime.h"
/**
 *  Project: HipGISAXS (High-Performance GISAXS)
 *
 *  File: ff_ana_gpu.cu
 *  Created: Oct 16, 2012
 *  Modified: Wed 08 Oct 2014 12:17:47 PM PDT
 *
 *  Author: Abhinav Sarje <asarje@lbl.gov>
 *  Developers: Slim Chourou <stchourou@lbl.gov>
 *              Abhinav Sarje <asarje@lbl.gov>
 *              Elaine Chan <erchan@lbl.gov>
 *              Alexander Hexemer <ahexemer@lbl.gov>
 *              Xiaoye Li <xsli@lbl.gov>
 *
 *  Licensing: The HipGISAXS software is only available to be downloaded and
 *  used by employees of academic research institutions, not-for-profit
 *  research laboratories, or governmental research facilities. Please read the
 *  accompanying LICENSE file before downloading the software. By downloading
 *  the software, you are agreeing to be bound by the terms of this
 *  NON-COMMERCIAL END USER LICENSE AGREEMENT.
 */

#include <iostream>
#include <fstream>
#include <complex>
#include <hip/hip_complex.h>
#include <stdio.h>

#include <ff/gpu/ff_ana_gpu.cuh>
#include <common/enums.hpp>
#include <common/constants.hpp>
#include <numerics/gpu/cu_complex_numeric.cuh>
#include <utils/gpu/cu_utilities.cuh>


namespace hig {

  extern __constant__ real_t tau_d;
  extern __constant__ real_t eta_d;
  extern __constant__ real_t transvec_d[3];

  /** Form Factor of Cube:
   *  L : (real) Length of the cube 
   *  q : (complex) q-vector
   *  ff = L^3  exp(j * qz * L/2) * sinc(qx * L/2) * sinc (qy * L/2) * sinc(qz * L/2)
   */
  __device__  __inline__ cucomplex_t FormFactorCube(cucomplex_t qx, cucomplex_t qy, cucomplex_t qz, 
          real_t L){
    cucomplex_t temp1 = cuCsinc (0.5 * qx * L);
    cucomplex_t temp2 = cuCsinc (0.5 * qy * L);
    cucomplex_t temp3 = cuCsinc (0.5 * qz * L);
    cucomplex_t temp4 = L * L * L * cuCexpi(0.5 * qz * L);
    return (temp1 * temp2 * temp3 * temp4);
  }
 
  __global__ void ff_cube_kernel (unsigned int nqy, unsigned int nqz, 
          real_t * qx, real_t * qy, cucomplex_t * qz, cucomplex_t * ff,
          RotMatrix_t rot,
          int nx, real_t * x, real_t * distr_x) {
    int i_z = blockDim.x * blockIdx.x + threadIdx.x;
    if (i_z < nqz){
      int i_y = i_z % nqy;
      cucomplex_t c_neg_unit = make_cuC(REAL_ZERO_, REAL_MINUS_ONE_);
      cucomplex_t mqx, mqy, mqz;
      rot.rotate(qx[i_y], qy[i_y], qz[i_z], mqx, mqy, mqz);
      cucomplex_t temp_ff = make_cuC(REAL_ZERO_, REAL_ZERO_);
      for (int i = 0; i < nx; i++){
        temp_ff = temp_ff + distr_x[i] * FormFactorCube(mqx, mqy, mqz, x[i]); 
      }
      cucomplex_t temp1 = transvec_d[0] * mqx + transvec_d[1] * mqy + transvec_d[2] * mqz;
      ff[i_z] =  temp_ff * cuCexpi(temp1);
    }
  } // ff_cube_kernel()

   
  bool AnalyticFormFactorG::compute_cube(const real_t tau, const real_t eta,
                  const std::vector<real_t>& x,
                  const std::vector<real_t>& distr_x,
                  const RotMatrix_t & rot, const std::vector<real_t>& transvec,
                  std::vector<complex_t>& ff) {
    unsigned int n_x = x.size(), n_distr_x = distr_x.size();
    const real_t *x_h = x.empty() ? NULL : &*x.begin();
    const real_t *distr_x_h = distr_x.empty() ? NULL : &*distr_x.begin();
    real_t transvec_h[3] = {transvec[0], transvec[1], transvec[2]};

    // construct device buffers
    real_t *x_d, *distr_x_d;

    hipMalloc((void**) &x_d, n_x * sizeof(real_t));
    hipMalloc((void**) &distr_x_d, n_distr_x * sizeof(real_t));

    // copy data to device buffers
    hipMemcpy(x_d, x_h, n_x * sizeof(real_t), hipMemcpyHostToDevice);
    hipMemcpy(distr_x_d, distr_x_h, n_distr_x * sizeof(real_t), hipMemcpyHostToDevice);

    //run_init(rot_h, transvec);
    hipMemcpyToSymbol(HIP_SYMBOL(transvec_d), transvec_h, 3*sizeof(real_t), 0, hipMemcpyHostToDevice); 

    int num_threads = 256;
    int num_blocks =  nqz_ / num_threads + 1;
    dim3 ff_grid_size(num_blocks, 1, 1);
    dim3 ff_block_size(num_threads, 1, 1);

    // the kernel
    ff_cube_kernel <<<num_blocks, num_threads >>> (nqy_, nqz_, 
            qx_, qy_, qz_, ff_, rot, n_x, x_d, distr_x_d);
    
    construct_output_ff(ff);

    hipFree(distr_x_d);
    hipFree(x_d);
   
    return true;
  } // AnalyticFormFactorG::compute_cube()

} // namespace hig
